#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>

__global__ void kernel(float * a , int offset)
{
  float x, c, s;
  int i = offset + threadIdx.x+ blockIdx.x * blockDim.x;
  x = 1.0 * i;
  s = sin(x);
  c = cos(x);
  a[i] += sqrt(s*s + c*c);
}

int main()
{
  constexpr int nStreams = 4;
  const int blockSize = 256;
  const int n = 4 * 1024 * blockSize * nStreams;
  const size_t sizeTot = sizeof(float) * n;
  const int streamSize = n / nStreams;

  float * aPinned;
  float * aDevice;
  
  hipStream_t streams[nStreams];

  checkCudaErrors(hipMalloc(&aDevice, sizeTot));
  checkCudaErrors(hipHostAlloc(&aPinned, sizeTot, hipHostMallocDefault));

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(int i = 0; i < nStreams ; ++i)
     checkCudaErrors(hipStreamCreate(&streams[i]));

  // sequential
  for(int i = 0; i < n ; ++i) { aPinned[i] = 0.0;}
  checkCudaErrors(hipEventRecord(start, 0));

  checkCudaErrors(hipMemcpy(aDevice, aPinned, sizeTot, hipMemcpyHostToDevice));
  kernel<<<n/blockSize, blockSize>>>(aDevice, 0);
  checkCudaErrors(hipMemcpy(aPinned, aDevice, sizeTot, hipMemcpyDeviceToHost));

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));

  float time;
  checkCudaErrors(hipEventElapsedTime(&time, start, stop));
  float maxVal = 0.0;
  for(int i = 0; i < n ; ++i) { maxVal = std::max(fabsf(aPinned[i] -1.0), maxVal);}
  printf("Time for sequential %fms, error = %e\n", time, maxVal);
 
  // Asynchronous
  for(int i = 0; i < n ; ++i) { aPinned[i] = 0.0;}
  checkCudaErrors(hipEventRecord(start, 0));

  for(int s = 0 ; s < nStreams; ++s) 
  {
    int offset = s * streamSize;
    checkCudaErrors(hipMemcpyAsync(aDevice + offset, aPinned + offset, streamSize * sizeof(float), 
                    hipMemcpyHostToDevice, streams[s]));
    kernel<<<streamSize/blockSize, blockSize, 0, streams[s]>>>(aDevice, offset);
    checkCudaErrors(hipMemcpyAsync(aPinned + offset, aDevice + offset, streamSize * sizeof(float), 
                    hipMemcpyDeviceToHost, streams[s]));
  }

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&time, start, stop));
  
  maxVal = 0.0;
  for(int i = 0; i < n ; ++i) { maxVal = std::max(fabsf(aPinned[i] -1.0), maxVal);}
  printf("Time for asynchronous %fms, error = %e\n", time, maxVal);
 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipHostFree(aPinned);

}
