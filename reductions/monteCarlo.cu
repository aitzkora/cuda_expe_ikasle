#include "hip/hip_runtime.h"
// adapted from CUDA fortran for Scientists and Engineers
#include <cstdio>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "hiprand.h"
typedef float scalar;


__global__ void finalSum(int * partial, int * total)
{
  extern __shared__ int psum[];
  int i = threadIdx.x;
  psum[i] = partial[i];
  __syncthreads();
  int  iNext = blockDim.x/2;
  while (iNext > 0)
  {
    if (i < iNext)
      psum[i] += psum[i+iNext];
    iNext >>= 1;
    __syncthreads();
  }
  if (i == 0) *total = psum[0];
}

__global__ void partialSum(scalar * input, int * partial, const int N) 
{
  extern __shared__ int psum[];
  int idX  = threadIdx.x + blockDim.x * blockIdx.x;
  int interior = 0;
  for (int i = idX; i < N ; i+= gridDim.x * blockDim.x)
    if ((input[i]*input[i]+input[i+N]*input[i+N]) <= 1.0)
       interior++;
  idX = threadIdx.x;
  psum[idX] = interior;
  __syncthreads();
  int iNext = blockDim.x / 2;
  while (iNext > 0)
  {
    if (idX < iNext)
      psum[idX] += psum[idX+iNext];
    iNext >>= 1;
    __syncthreads();
  }
  if (idX == 0) partial[blockIdx.x] = psum[0];
}

__global__ void partialSumDiverge(scalar *input, int * partial, const int N)
{
}

__global__ void partialSumDummy(scalar *input, int * partial, const int N)
{
}

scalar computeSum(const scalar * XY, int N)
{
  int interior = 0;
  for(int i = 0; i < N; ++i)
  {
    if ((XY[i] * XY[i] + XY[i+N]*XY[i+N]) <= static_cast<scalar>(1.0))
       interior++;
  }
  return interior/static_cast<scalar>(N);
}

#define BLOCK_SIZE 512
#define NB_BLOCKS 256
#define NB_KERNELS 3
typedef void(*KernelPtr) (scalar *, int * partial, const int N);
KernelPtr kernels[NB_KERNELS] = {&partialSum, &partialSumDiverge, partialSumDummy};
const char * kernelNames[NB_KERNELS]= {"noDiverging", "diverging", "oneThread" };

int main(int argc, char * argv[])
{
  const int N = (argc < 2) ? 4 * NB_BLOCKS * BLOCK_SIZE  : atoi(argv[1]);
  const int twoN = N << 1;

  int * partial;
  hipMalloc(&partial, NB_BLOCKS * sizeof(int));
  int * interiorGPU;
  hipHostAlloc(&interiorGPU, sizeof(int), hipHostMallocMapped);
  // xy data on host;
  scalar * xy = (scalar*) malloc(twoN * sizeof(scalar));
  


  // xy data on Device
  scalar * xyDevice;
  size_t sizeXY = twoN * sizeof(scalar);
  checkCudaErrors(hipMalloc(&xyDevice, sizeXY));

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed  = 1234;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, xyDevice, twoN);

  checkCudaErrors(hipMemcpy(xy, xyDevice, sizeXY, hipMemcpyDeviceToHost));

  scalar res = computeSum(xy, N);

  partialSum<<<NB_BLOCKS,BLOCK_SIZE,BLOCK_SIZE*sizeof(int)>>>(xyDevice, partial, N);
  checkCudaErrors(hipGetLastError());
  finalSum<<<1,NB_BLOCKS,NB_BLOCKS*sizeof(int)>>>(partial, interiorGPU);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  
  scalar resGPU = *interiorGPU/static_cast<scalar>(N);
    if (fabsf(resGPU-res) > 1.e-6)
  {
    fprintf(stderr, "********* ERROR ********** \n bad kernel computation\n");
  }
  else
  { 
    printf("********* SUCCESS *********** \nN = %d, π ~ %f\n", N ,4.0 * resGPU);
  }

  // chrono des reductions
  
  //for(int ker = 0; ker < 3; ++ker) decommenter ici quand les autres noyaux sont implementes
  for(int ker = 0; ker < 1; ++ker) 
  {
    hipEvent_t start, stop;
    float timeInMs;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));
    int nbTries = 10;
    for(int j = 0; j < nbTries ; ++j) {
       kernels[ker]<<<NB_BLOCKS,BLOCK_SIZE,BLOCK_SIZE*sizeof(int)>>>(xyDevice, partial, N);
    }
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&timeInMs, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    printf("ker = %s, time = %f\n", kernelNames[ker], timeInMs / nbTries);
  } 

  checkCudaErrors(hipFree(xyDevice));
  checkCudaErrors(hipFree(partial));
  checkCudaErrors(hipHostFree(interiorGPU));
  free(xy);
  return 0;
}
